#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <string>
#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <windows.h>

using namespace std;

__global__ void matrixMult(const int* A, const int* B, int* C, int matrix)
{
    int i = matrix * (blockDim.y * blockIdx.y + threadIdx.y);
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int sum = 0;

    for (int k = 0; k < matrix; k++)
        sum += A[i + k] * B[k * matrix + j];

    int ind = matrix * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
    C[ind] = sum;
}

int main(int argc, char** argv) {

    SetConsoleOutputCP(CP_UTF8);


    int matrix = 9543;
    int threads = 16;

    size_t size = matrix * matrix * sizeof(int);

    int* a = (int*)malloc(size);
    int* b = (int*)malloc(size);
    int* c = (int*)malloc(size);

    for (int i = 0; i < matrix; i++) {
        for (int j = 0; j < matrix; j++) {
            a[i * matrix + j] = i * j;
            b[i * matrix + j] = i * j;
        }
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int* calcA = NULL;
    hipMalloc((void**)&calcA, size);

    int* calcB = NULL;
    hipMalloc((void**)&calcB, size);

    int* calcC = NULL;
    hipMalloc((void**)&calcC, size);

    hipMemcpy(calcA, a, size, hipMemcpyHostToDevice);
    hipMemcpy(calcB, b, size, hipMemcpyHostToDevice);

    cout << "Старт" << endl;

    dim3 threadsPerBlock = dim3(threads, threads);
    dim3 blocksPerGrid = dim3(matrix / threads, matrix / threads);

    hipEventRecord(start, 0);
    matrixMult << < blocksPerGrid, threadsPerBlock >> > (calcA, calcB, calcC, matrix);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float kernelTime;
    hipEventElapsedTime(&kernelTime, start, stop);
    double time = kernelTime;
    cout << " Количество потоков: " << threads
        << "; Время: " << kernelTime / 1000 << " секунд" << endl;
    hipMemcpy(c, calcC, size, hipMemcpyDeviceToHost);

    hipFree(calcA);
    hipFree(calcB);
    hipFree(calcC);
    free(a);
    free(b);
    free(c);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}